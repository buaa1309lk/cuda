#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#define datasize 10242048
int data[datasize];

void creatNum(int* data,int size){
	int i;
	for(i=0;i<size;i++)
		data[i]=rand()%10;
}


__global__ static void sumOfSquare(int *num,int *result){
	int sum=0;
	int i;
	for(i=0;i<datasize;i++)
		sum+=num[i]*num[i];
	*result=sum;
}


int main()
{
	using namespace std;

	creatNum(data,datasize);
	int *gpudata,*result;

	hipMalloc((void**)&gpudata,sizeof(int)*datasize);
	hipMalloc((void**)&result,sizeof(int));
	hipMemcpy(gpudata,data,sizeof(int)*datasize,hipMemcpyHostToDevice);
	sumOfSquare<<<1,1,0>>>(gpudata,result);

	int sum;
	hipMemcpy(&sum,result,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	cout<<sum<<endl;

	sum=0;
	int i;
	for(i=0;i<datasize;i++)
		sum+=data[i]*data[i];
	cout<<sum<<endl;

	return 0;
}

