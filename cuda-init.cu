#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;
bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);
	if(count==0)
	{
		cout<<"there is no device"<<endl;
		return 0;
	}

	int i;	
	for(i=0;i<count;i++)
	{
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop,i)==hipSuccess)
		{
			if(prop.major>=1)
				break;
		}
	}

	if(i==count){
		cout<<"There is no device supproting CUDA 1.X"<<endl;
		return 0;
	}

	hipSetDevice(i);
		return true;
}

int main()
{
	using namespace std;
	if(!InitCUDA())
		return 0;
	cout<<"CUDA INITIALIZED"<<endl;
	return 0;
}
