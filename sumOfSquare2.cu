#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#define datasize 10242048
int data[datasize];

void creatNum(int* data,int size){
	int i;
	for(i=0;i<size;i++)
		data[i]=rand()%10;
}


__global__ static void sumOfSquare(int *num,int *result,clock_t *time){
	int sum=0;
	int i;
	clock_t start=clock();
	for(i=0;i<datasize;i++)
		sum+=num[i]*num[i];
	*result=sum;
	*time=clock()-start;
}


int main()
{
	using namespace std;

	creatNum(data,datasize);
	int *gpudata,*result;
	clock_t *time;
	hipMalloc((void**)&gpudata,sizeof(int)*datasize);
	hipMalloc((void**)&result,sizeof(int));
	hipMalloc((void**)&time,sizeof(clock_t));
	hipMemcpy(gpudata,data,sizeof(int)*datasize,hipMemcpyHostToDevice);
	sumOfSquare<<<1,1,0>>>(gpudata,result,time);

	int sum;
	clock_t time_used;
	hipMemcpy(&sum,result,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&time_used,time,sizeof(clock_t),hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	cout<<"sum is:"<<sum<<"and time used is: "<<time_used<<endl;

	sum=0;
	clock_t cputime=clock();
	int i;
	for(i=0;i<datasize;i++)
		sum+=data[i]*data[i];
	time2=clock()-cputime;
	cout<<"cpu sum is:"<<sum<<"and time used is:"<<cputime<<endl;

	return 0;
}

