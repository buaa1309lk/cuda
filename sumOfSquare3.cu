#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#define datasize 10242048
#define THREAD_NUM 256


int data[datasize];

void creatNum(int* data,int size){
	int i;
	for(i=0;i<size;i++)
		data[i]=rand()%10;
}


__global__ static void sumOfSquare(int *num,int *result,clock_t *time){
	const int tid=threadIdx.x;
	const int size=datasize/THREAD_NUM;
		
	int sum=0;
	int i;
	clock_t start;
	if(tid==0)
		start=clock();
	for(i=tid*size;i<(tid+1)*size;i++)
		sum+=num[i]*num[i];
	result[tid]=sum;

	if(tid==0)
		*time=clock()-start;
}


int main()
{
	using namespace std;

	creatNum(data,datasize);
	int *gpudata,*result;
	clock_t *time;
	hipMalloc((void**)&gpudata,sizeof(int)*datasize);
	hipMalloc((void**)&result,sizeof(int)*THREAD_NUM);
	hipMalloc((void**)&time,sizeof(clock_t));
	hipMemcpy(gpudata,data,sizeof(int)*THREAD_NUM,hipMemcpyHostToDevice);
	sumOfSquare<<<1,THREAD_NUM,0>>>(gpudata,result,time);

	int sum[THREAD_NUM];
	clock_t time_used;
	hipMemcpy(&sum,result,sizeof(int)*THREAD_NUM,hipMemcpyDeviceToHost);
	hipMemcpy(&time_used,time,sizeof(clock_t),hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	
	int final_sum=0;
	int i;
	for(i=0;i<THREAD_NUM;i++)
		final_sum+=sum[i];
	cout<<"sum is:"<<final_sum<<"and time used is: "<<time_used<<endl;

	int	sum_cpu=0;
	clock_t cputime=clock();
	for(i=0;i<datasize;i++)
		sum_cpu+=data[i]*data[i];
	cputime=clock()-cputime;
	cout<<"cpu sum is:"<<sum_cpu<<"and time used is:"<<cputime<<endl;

	return 0;
}

